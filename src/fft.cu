#include "hip/hip_runtime.h"
// File: fft_async.cu
// Compile with:
//   nvcc -std=c++17 fft_async.cu -I/path/to/AudioFile -lcufft -o bin/fft_async

#define DR_WAV_IMPLEMENTATION
#include "AudioFile.h"

#include <iostream>
#include <string>
#include <vector>
#include <filesystem>
#include <fstream>
#include <cmath>
#include <numeric>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

namespace fs = std::filesystem;

// --- Helper Functions ---
inline void checkCuda(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " (" << hipGetErrorString(err) << ")\n";
        std::exit(EXIT_FAILURE);
    }
}

inline void checkCufft(hipfftResult err, const char *msg) {
    if (err != HIPFFT_SUCCESS) {
        std::cerr << "cuFFT Error: " << msg << " (" << err << ")\n";
        std::exit(EXIT_FAILURE);
    }
}

// --- Data Structure for each stream's resources ---
struct StreamData {
    hipStream_t stream;
    hipfftHandle plan;
    fs::path filePath;
    
    // Pinned host memory for async copies
    float* h_in_pinned;
    hipfftComplex* h_out_pinned;

    // Device memory
    float* d_in;
    hipfftComplex* d_out;

    // Audio metadata
    int numSamples;
    float sampleRate;
    bool inUse = false;
};

// --- CPU task to analyze results after GPU is done ---
// --- New Helper function to convert frequency to a musical note ---
std::string frequencyToNote(float freq) {
    if (freq <= 0) return "N/A";
    const char* noteNames[] = {"C", "C#", "D", "D#", "E", "F", "F#", "G", "G#", "A", "A#", "B"};
    
    // Calculate the MIDI note number
    int midiNote = round(12 * log2(freq / 440.0f) + 69);
    
    // Determine the note name and octave
    std::string noteName = noteNames[midiNote % 12];
    int octave = (midiNote / 12) - 1;
    
    return noteName + std::to_string(octave);
}
// --- CPU task to analyze results after GPU is done ---
void postProcess(StreamData& data, std::ofstream& resultsFile, int transformSize) {
    const int TOP_N = 3; // Find the top 3 frequencies
    const int fft_output_size = (data.numSamples / 2) + 1;
    
    // Store all frequency magnitudes and indices
    std::vector<std::pair<float, int>> magnitudes;
    magnitudes.reserve(fft_output_size);

    for (int i = 1; i < fft_output_size; ++i) { // Start at 1 to ignore DC offset
        float mag = sqrtf(data.h_out_pinned[i].x * data.h_out_pinned[i].x + data.h_out_pinned[i].y * data.h_out_pinned[i].y);
        magnitudes.push_back({mag, i});
    }

    // Sort to find the frequencies with the highest magnitude
    std::sort(magnitudes.begin(), magnitudes.end(), [](const auto& a, const auto& b) {
        return a.first > b.first;
    });

    std::cout << "  > Finished " << data.filePath.filename().string() << ":\n";
    resultsFile << data.filePath.filename().string();

    // Get the top N results
    for (int i = 0; i < TOP_N; ++i) {
        if (i < magnitudes.size()) {
            int peakIndex = magnitudes[i].second;
            float peakFrequency = static_cast<float>(peakIndex) * data.sampleRate / transformSize;
            std::string note = frequencyToNote(peakFrequency);

            std::cout << "    - Freq " << i + 1 << ": " << peakFrequency << " Hz (" << note << ")\n";
            resultsFile << "," << peakFrequency << "," << note;
        } else {
            resultsFile << ",,"; // Add empty columns if fewer than N peaks are found
        }
    }
    resultsFile << "\n";
    data.inUse = false; // Mark stream as free
}
// --- Main Application ---
int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <input_dir> <output_dir>\n";
        return 1;
    }

    fs::path inDir = argv[1];
    fs::path outDir = argv[2];
    fs::create_directories(outDir);

    // --- Collect all WAV files ---
    std::vector<fs::path> filesToProcess;
    for (const auto& entry : fs::directory_iterator(inDir)) {
        if (entry.is_regular_file() && entry.path().extension() == ".wav") {
            filesToProcess.push_back(entry.path());
        }
    }
    if (filesToProcess.empty()) {
        std::cout << "No .wav files found in input directory.\n";
        return 0;
    }
    int totalFiles = filesToProcess.size();

    // --- Setup Asynchronous Pipeline ---
    const int NUM_STREAMS = 4;
    std::vector<StreamData> streamData(NUM_STREAMS);

    const int MAX_SAMPLES = 44100 * 60; // Max 60 seconds at 44.1kHz
    const int MAX_FFT_OUTPUTS = MAX_SAMPLES / 2 + 1;

    // --- Initialization Loop (with corrected plan creation) ---
    for (int i = 0; i < NUM_STREAMS; ++i) {
        checkCuda(hipStreamCreate(&streamData[i].stream), "Stream Create");
        checkCufft(hipfftCreate(&streamData[i].plan), "hipfftCreate");
        
        // Configure the plan ONCE for the maximum size
        size_t workSize;
        checkCufft(hipfftMakePlan1d(streamData[i].plan, MAX_SAMPLES, HIPFFT_R2C, 1, &workSize), "Make plan for max size");
        
        checkCuda(hipHostMalloc(&streamData[i].h_in_pinned, MAX_SAMPLES * sizeof(float)), "MallocHost h_in");
        checkCuda(hipHostMalloc(&streamData[i].h_out_pinned, MAX_FFT_OUTPUTS * sizeof(hipfftComplex)), "MallocHost h_out");
        
        checkCuda(hipMalloc(&streamData[i].d_in, MAX_SAMPLES * sizeof(float)), "Malloc d_in");
        checkCuda(hipMalloc(&streamData[i].d_out, MAX_FFT_OUTPUTS * sizeof(hipfftComplex)), "Malloc d_out");
    }

    std::ofstream resultsFile(outDir / "frequency_results.csv");
    resultsFile << "filename,freq1_hz,note1,freq2_hz,note2,freq3_hz,note3\n";

    int fileIndex = 0;
    int processedCount = 0;
    std::cout << "Starting processing for " << totalFiles << " files with " << NUM_STREAMS << " streams...\n";

    // --- Main Processing Loop ---
    while (processedCount < totalFiles) {
        for (int i = 0; i < NUM_STREAMS; ++i) {
            // Check if a previously launched stream is done
            if (streamData[i].inUse && hipStreamQuery(streamData[i].stream) == hipSuccess) {
                postProcess(streamData[i], resultsFile, MAX_SAMPLES); // Pass MAX_SAMPLES
                processedCount++;
            }

            // If the stream is free and there are files left, launch a new one
            if (!streamData[i].inUse && fileIndex < totalFiles) {
                // Load audio file (CPU task)
                AudioFile<float> audioFile;
                if (!audioFile.load(filesToProcess[fileIndex])) {
                    std::cerr << "ERROR: Failed to load " << filesToProcess[fileIndex].filename().string() << ", skipping.\n";
                    fileIndex++;
                    processedCount++;
                    continue;
                }
                
                int currentSamples = audioFile.getNumSamplesPerChannel();
                if (currentSamples > MAX_SAMPLES) {
                    std::cerr << "ERROR: File " << filesToProcess[fileIndex].filename().string() << " is too large, skipping.\n";
                    fileIndex++;
                    processedCount++;
                    continue;
                }

                // Prepare data for this stream
                auto& currentStream = streamData[i];
                currentStream.inUse = true;
                currentStream.filePath = filesToProcess[fileIndex];
                currentStream.numSamples = currentSamples;
                currentStream.sampleRate = audioFile.getSampleRate();
                std::copy(audioFile.samples[0].begin(), audioFile.samples[0].end(), currentStream.h_in_pinned);
                
                // --- Launch Async GPU Pipeline (plan is already made) ---
                checkCufft(hipfftSetStream(currentStream.plan, currentStream.stream), "hipfftSetStream");
                
                checkCuda(hipMemcpyAsync(currentStream.d_in, currentStream.h_in_pinned, currentStream.numSamples * sizeof(float), hipMemcpyHostToDevice, currentStream.stream), "MemcpyAsync H2D");
                checkCufft(hipfftExecR2C(currentStream.plan, currentStream.d_in, currentStream.d_out), "hipfftExecR2C");
                checkCuda(hipMemcpyAsync(currentStream.h_out_pinned, currentStream.d_out, (currentStream.numSamples / 2 + 1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost, currentStream.stream), "MemcpyAsync D2H");
                
                fileIndex++;
            }
        }
    }
    
    // --- Final Synchronization and Cleanup ---
    checkCuda(hipDeviceSynchronize(), "Final Device Sync");
    
    // Inside the final cleanup loop at the end of main
    for (int i = 0; i < NUM_STREAMS; ++i) {
        if (streamData[i].inUse) {
            postProcess(streamData[i], resultsFile, MAX_SAMPLES); // Pass MAX_SAMPLES
        }
    }
        
    resultsFile.close();

    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipfftDestroy(streamData[i].plan);
        hipStreamDestroy(streamData[i].stream);
        hipHostFree(streamData[i].h_in_pinned);
        hipHostFree(streamData[i].h_out_pinned);
        hipFree(streamData[i].d_in);
        hipFree(streamData[i].d_out);
    }
    
    std::cout << "Done. All files processed.\n";
    return 0;
}